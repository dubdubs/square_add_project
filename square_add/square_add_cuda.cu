#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void square_add_kernel(const float* a, const float* b, float* out, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        out[i] = a[i] * a[i] + b[i] * b[i];
    }
}

at::Tensor square_add_cuda(const at::Tensor& a, const at::Tensor& b) {
    auto out = torch::zeros_like(a);
    int size = a.numel();
    int threads = 256;
    int blocks = (size + threads - 1) / threads;

    square_add_kernel<<<blocks, threads>>>(
        a.data_ptr<float>(), b.data_ptr<float>(), out.data_ptr<float>(), size);

    return out;
}